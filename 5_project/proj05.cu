#include "hip/hip_runtime.h"
﻿
// Monte Carlo simulation of golf balls:

// system includes
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <assert.h>
#include <malloc.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	131072
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE	64     // number of threads per block
#endif

#define NUMBLOCKS	( NUMTRIALS / BLOCKSIZE )


#define CSV

#define GRAVITY		32.2f

const float BEFOREY = 70.f;
const float AFTERY  = 10.f;
const float DISTX   = 50.f;

__device__ const float RADIUS   =    5.f;		// so that the device (GPU) can see this variable

const float BEFOREYDY = 5.f;
const float AFTERYDY  = 1.f;
const float DISTXDX   = 5.f;

float	hbeforey[NUMTRIALS];
float	haftery[NUMTRIALS];
float	hdistx[NUMTRIALS];
int	hsuccesses[NUMTRIALS];


float
Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}


// call this if you want to force your program to use
// a different random number sequence every time you run it:
void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
	srand(seed);
}


void
CudaCheckError()
{
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		fprintf(stderr, "Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	}
}


__device__
float
Sqr(float x)
{
	return x * x;
}


__device__
float
Length(float dx, float dy)
{
	return  sqrt(Sqr(dx) + Sqr(dy));
}


#define IN
#define OUT

__global__
void
MonteCarlo( IN float* dbeforey, IN float* daftery, IN float* ddistx, OUT int* dsuccesses)
{
	//unsigned int numItems = blockDim.x;		// don't need this for this project
	//unsigned int wgNum    = blockIdx.x;		// don't need this for this project
	//unsigned int tnum     = threadIdx.x;		// don't need this for this project
	// "gid", the global identifier, is essentially the for-loop index:
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

	dsuccesses[gid] = 0;

	// randomize everything:

	float beforey = dbeforey[gid];
	float aftery  = daftery[gid];
	float distx   = ddistx[gid];

	float vx = sqrt(2.f * GRAVITY * (beforey - aftery));
	float t  = sqrt((2.f * aftery) / GRAVITY);
	float x  = vx * t;
	if (fabs(x - distx) <= RADIUS)
		dsuccesses[gid] = 1;
}


int
main(int argc, char* argv[])
{
	TimeOfDaySeed();               // seed the random number generator

	// fill the random-value arrays:
	for (int n = 0; n < NUMTRIALS; n++)
	{
		hbeforey[n] = Ranf(BEFOREY - BEFOREYDY, BEFOREY + BEFOREYDY);
		haftery[n]  = Ranf(AFTERY - AFTERYDY,   AFTERY + AFTERYDY);
		hdistx[n]   = Ranf(DISTX - DISTXDX,     DISTX + DISTXDX);
	}

	// allocate device memory:

	float* dbeforey, * daftery, * ddistx;
	int*   dsuccesses;

	hipMalloc((void**)(&dbeforey), NUMTRIALS * sizeof(float));
	hipMalloc((void**)(&daftery),  NUMTRIALS * sizeof(float));
	hipMalloc((void**)(&ddistx),   NUMTRIALS * sizeof(float));

	hipMalloc((void**)(&dsuccesses), NUMTRIALS * sizeof(int));

	CudaCheckError();


	// copy host memory to the device:

	hipMemcpy(dbeforey, hbeforey, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(daftery,  haftery,  NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(ddistx,   hdistx,   NUMTRIALS * sizeof(float),   hipMemcpyHostToDevice);

	CudaCheckError();


	// setup the execution parameters:
	dim3 threads(BLOCKSIZE, 1, 1);
	dim3 grid(NUMBLOCKS, 1, 1);

	// create and start timer
	hipDeviceSynchronize();

	// allocate CUDA events that we'll use for timing:
	hipEvent_t start, stop;
	hipEventCreate(&start);
	CudaCheckError();
	hipEventCreate(&stop);
	CudaCheckError();

	// record the start event:
	hipEventRecord(start, NULL);
	CudaCheckError();

	// execute the kernel:
	MonteCarlo<<< grid, threads >>>( dbeforey, daftery, ddistx, dsuccesses );

	// record the stop event:
	hipEventRecord(stop, NULL);

	// wait for the stop event to complete:
	hipEventSynchronize(stop);

	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);
	CudaCheckError();

	// copy result from the device to the host:
	hipMemcpy(hsuccesses, dsuccesses, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost);
	CudaCheckError();

	// compute the sum :
	int numSuccesses = 0;
	for (int i = 0; i < NUMTRIALS; i++)
	{
		numSuccesses += hsuccesses[i];
	}

	float probability = (float)numSuccesses / (float)NUMTRIALS;

	// compute and print the performance:
	double secondsTotal    = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;

#ifdef CSV
	fprintf(stderr, "%10d , %8d , %10.4lf , %6.2f\n",
		NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, 100.f * probability);
#else
	fprintf(stderr, "Number of Trials = %10d, Blocksize = %8d, MegaTrials/Second = %10.4lf, Probability = %6.2f%%\n",
		NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, 100. * probability);
#endif

	// clean up device memory:
	hipFree(dbeforey);
	hipFree(daftery);
	hipFree(ddistx);
	hipFree(dsuccesses);

	CudaCheckError();

	return 0;
}
